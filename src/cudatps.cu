#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "cudatps.h"

#define MAXTHREADPBLOCK 1024

// Kernel definition
__global__ void tpsCuda(double* cudaImageCoord, int width, int height, float* solution, float* cudaKeyCol, float* cudaKeyRow, uint numOfKeys)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  double newCoord = solution[0] + x*solution[1] + y*solution[2];

  for (uint i = 0; i < numOfKeys; i++) {
    double r = (x-cudaKeyCol[i])*(x-cudaKeyCol[i]) + (y-cudaKeyRow[i])*(y-cudaKeyRow[i]);
    if (r != 0.0) newCoord += r*log(r) * solution[i+3];
  }
  if (x*height+y < width*height)
    cudaImageCoord[x*height+y] = newCoord;
}

void tps::CudaTPS::callKernel(float *cudaSolution, double *imageCoord, dim3 threadsPerBlock, dim3 numBlocks) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  tpsCuda<<<numBlocks, threadsPerBlock>>>(cudaImageCoord, width, height, cudaSolution, cudaKeyCol, cudaKeyRow, targetKeypoints_.size());
  hipDeviceSynchronize(); 
  hipMemcpy(imageCoord, cudaImageCoord, width*height*sizeof(double), hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "Time = " << elapsedTime << " ms\n";
}

void tps::CudaTPS::run() {
	allocResources();
	allocCudaResources();

  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(std::ceil(1.0*width/threadsPerBlock.x), std::ceil(1.0*height/threadsPerBlock.y));

  callKernel(cudaSolutionCol, imageCoordCol, threadsPerBlock, numBlocks);
  callKernel(cudaSolutionRow, imageCoordRow, threadsPerBlock, numBlocks);

  // std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

  for (int col = 0; col < width; col++)
    for (int row = 0; row < height; row++) {
      double newCol = imageCoordCol[col*height+row];
      double newRow = imageCoordRow[col*height+row];
      int value = targetImage_.bilinearInterpolation(newCol, newRow);
      registredImage.changePixelAt(col, row, value);
    }
  registredImage.save(outputName_);

	freeResources();
	freeCudaResources();

	hipDeviceReset();
}

void tps::CudaTPS::allocResources() {
  imageCoordCol = (double*)malloc(width*height*sizeof(double));
  imageCoordRow = (double*)malloc(width*height*sizeof(double));
  createCudaSolution();
  createCudaKeyPoint();
}

void tps::CudaTPS::createCudaSolution() {
  std::vector<float> solutionCol;
  std::vector<float> solutionRow;
  cudalienarSolver.solveLinearSystems();
  solutionCol = cudalienarSolver.getSolutionCol();
  solutionRow = cudalienarSolver.getSolutionRow();
  floatSolCol = (float*)malloc((targetKeypoints_.size()+3)*sizeof(float));
  floatSolRow = (float*)malloc((targetKeypoints_.size()+3)*sizeof(float));
  for (uint i = 0; i < (targetKeypoints_.size()+3); i++) {
    floatSolCol[i] = solutionCol[i];
    floatSolRow[i] = solutionRow[i];
  }
}

void tps::CudaTPS::createCudaKeyPoint() {
  floatKeyCol = (float*)malloc(targetKeypoints_.size()*sizeof(float));
  floatKeyRow = (float*)malloc(targetKeypoints_.size()*sizeof(float));
  for (uint i = 0; i < targetKeypoints_.size(); i++) {
    floatKeyCol[i] = referenceKeypoints_[i].x;
    floatKeyRow[i] = referenceKeypoints_[i].y;
  }
}

void tps::CudaTPS::allocCudaResources() {
  hipMalloc(&cudaImageCoord, width*height*sizeof(double));
  hipMalloc(&cudaSolutionCol, (targetKeypoints_.size()+3)*sizeof(float));
  hipMalloc(&cudaSolutionRow, (targetKeypoints_.size()+3)*sizeof(float));
  hipMemcpy(cudaSolutionCol, floatSolCol, (targetKeypoints_.size()+3)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaSolutionRow, floatSolRow, (targetKeypoints_.size()+3)*sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&cudaKeyCol, targetKeypoints_.size()*sizeof(float));
  hipMalloc(&cudaKeyRow, targetKeypoints_.size()*sizeof(float));
  hipMemcpy(cudaKeyCol, floatKeyCol, targetKeypoints_.size()*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaKeyRow, floatKeyRow, targetKeypoints_.size()*sizeof(float), hipMemcpyHostToDevice);
}

void tps::CudaTPS::freeResources() {
  free(imageCoordCol);
  free(imageCoordRow);
  free(floatSolCol);
  free(floatSolRow);
  free(floatKeyCol);
  free(floatKeyRow);
}

void tps::CudaTPS::freeCudaResources() {
  hipFree(cudaImageCoord);
  hipFree(cudaSolutionCol);
  hipFree(cudaSolutionRow);
  hipFree(cudaKeyCol);
  hipFree(cudaKeyRow);
	hipDeviceSynchronize();
}
