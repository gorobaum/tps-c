#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "cudatps.h"

#define MAXTHREADPBLOCK 1024

// Kernel definition
__global__ void tpsCuda(double* cudaImageCoord, int width, int heigth, float* solution, float* cudaKeyX, float* cudaKeyY, uint numOfKeys)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  double newCoord = solution[0] + x*solution[1] + y*solution[2];

  for (uint i = 0; i < numOfKeys; i++) {
    double r = (x-cudaKeyX[i])*(x-cudaKeyX[i]) + (y-cudaKeyY[i])*(y-cudaKeyY[i]);
    newCoord += r*log(r) * solution[i+3];
  }
  if (x*width+y < width*heigth)
    cudaImageCoord[x*width+y] = newCoord;
}

void tps::CudaTPS::callKernel(float *cudaSolution, double *imageCoord, dim3 threadsPerBlock, dim3 numBlocks) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  tpsCuda<<<numBlocks, threadsPerBlock>>>(cudaImageCoord, dimensions[1], dimensions[0], cudaSolution, cudaKeyX, cudaKeyY, targetKeypoints_.size());
  hipDeviceSynchronize(); 
  hipMemcpy(imageCoord, cudaImageCoord, dimensions[0]*dimensions[1]*sizeof(double), hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "Time = " << elapsedTime << " ms\n";
}

void tps::CudaTPS::run() {
	dimensions = registredImage.getDimensions(); 
  findSolutions();
	allocResources();
	allocCudaResources();

  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(std::ceil(1.0*dimensions[0]/threadsPerBlock.x), std::ceil(1.0*dimensions[1]/threadsPerBlock.y));

  callKernel(cudaSolutionX, imageCoordX, threadsPerBlock, numBlocks);
  callKernel(cudaSolutionY, imageCoordY, threadsPerBlock, numBlocks);

  std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

  for (int x = 0; x < dimensions[0]; x++)
    for (int y = 0; y < dimensions[1]; y++) {
      double newX = imageCoordX[x*dimensions[1]+y];
      double newY = imageCoordY[x*dimensions[1]+y];
      uchar value = targetImage_.bilinearInterpolation<uchar>(newX, newY);
      registredImage.changePixelAt(x, y, value);
    }
  registredImage.save();

	freeResources();
	freeCudaResources();

	hipDeviceReset();
}

void tps::CudaTPS::allocResources() {
  imageCoordX = (double*)malloc(dimensions[0]*dimensions[1]*sizeof(double));
  imageCoordY = (double*)malloc(dimensions[0]*dimensions[1]*sizeof(double));
  createCudaSolution();
  createCudaKeyPoint();
}

void tps::CudaTPS::createCudaSolution() {
  floatSolX = (float*)malloc((targetKeypoints_.size()+3)*sizeof(float));
  floatSolY = (float*)malloc((targetKeypoints_.size()+3)*sizeof(float));
  for (uint i = 0; i < (targetKeypoints_.size()+3); i++) {
    floatSolX[i] = solutionX.at<float>(i);
    floatSolY[i] = solutionY.at<float>(i);
  }
}

void tps::CudaTPS::createCudaKeyPoint() {
  floatKeyX = (float*)malloc(targetKeypoints_.size()*sizeof(float));
  floatKeyY = (float*)malloc(targetKeypoints_.size()*sizeof(float));
  for (uint i = 0; i < targetKeypoints_.size(); i++) {
    floatKeyX[i] = referenceKeypoints_[i].x;
    floatKeyY[i] = referenceKeypoints_[i].y;
  }
}

void tps::CudaTPS::allocCudaResources() {
  hipMalloc(&cudaImageCoord, dimensions[0]*dimensions[1]*sizeof(double));
  hipMalloc(&cudaSolutionX, (targetKeypoints_.size()+3)*sizeof(float));
  hipMalloc(&cudaSolutionY, (targetKeypoints_.size()+3)*sizeof(float));
  hipMemcpy(cudaSolutionX, floatSolX, (targetKeypoints_.size()+3)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaSolutionY, floatSolY, (targetKeypoints_.size()+3)*sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&cudaKeyX, targetKeypoints_.size()*sizeof(float));
  hipMalloc(&cudaKeyY, targetKeypoints_.size()*sizeof(float));
  hipMemcpy(cudaKeyX, floatKeyX, targetKeypoints_.size()*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaKeyY, floatKeyY, targetKeypoints_.size()*sizeof(float), hipMemcpyHostToDevice);
}

void tps::CudaTPS::freeResources() {
  free(imageCoordX);
  free(imageCoordY);
  free(floatSolX);
  free(floatSolY);
  free(floatKeyX);
  free(floatKeyY);
}

void tps::CudaTPS::freeCudaResources() {
  hipFree(cudaImageCoord);
  hipFree(cudaSolutionX);
  hipFree(cudaSolutionY);
  hipFree(cudaKeyX);
  hipFree(cudaKeyY);
	hipDeviceSynchronize();
}
