#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

#include "cudatps.h"

#define MAXTHREADPBLOCK 1024

// Kernel definition
__device__ double cudaGetPixel(int x, int y, uchar* image, int width, int height) {
  if (x > width-1 || x < 0) return 0;
  if (y > height-1 || y < 0) return 0;
  return image[x*height+y];
}

// Kernel definition
__device__ double cudaBilinearInterpolation(double col, double row, uchar* image, int width, int height) {
  int u = trunc(col);
  int v = trunc(row);

  uchar pixelOne = cudaGetPixel(u, v, image, width, height);
  uchar pixelTwo = cudaGetPixel(u+1, v, image, width, height);
  uchar pixelThree = cudaGetPixel(u, v+1, image, width, height);
  uchar pixelFour = cudaGetPixel(u+1, v+1, image, width, height);

  double interpolation = (u+1-col)*(v+1-row)*pixelOne
                        + (col-u)*(v+1-row)*pixelTwo 
                        + (u+1-col)*(row-v)*pixelThree
                        + (col-u)*(row-v)*pixelFour;
  return interpolation;
}

// Kernel definition
__global__ void cudaRegistredImage(double* cudaImageCoordX, double* cudaImageCoordY, uchar* cudaImage, uchar* cudaRegImage, int width, int height) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;

  double newX = cudaImageCoordX[x*height+y];
  double newY = cudaImageCoordY[x*height+y];
  cudaRegImage[x*height+y] = cudaBilinearInterpolation(newX, newY, cudaImage, width, height);
}

// Kernel definition
__global__ void tpsCuda(double* cudaImageCoord, int width, int height, float* solution, float* cudaKeyCol, float* cudaKeyRow, uint numOfKeys)
{
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  double newCoord = solution[0] + x*solution[1] + y*solution[2];

  for (uint i = 0; i < numOfKeys; i++) {
    double r = (x-cudaKeyCol[i])*(x-cudaKeyCol[i]) + (y-cudaKeyRow[i])*(y-cudaKeyRow[i]);
    if (r != 0.0) newCoord += r*log(r) * solution[i+3];
  }
  if (x*height+y < width*height)
    cudaImageCoord[x*height+y] = newCoord;
}

void tps::CudaTPS::callKernel(double *cudaImageCoord, float *cudaSolution, dim3 threadsPerBlock, dim3 numBlocks) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  tpsCuda<<<numBlocks, threadsPerBlock>>>(cudaImageCoord, width, height, cudaSolution, cudaKeyCol, cudaKeyRow, targetKeypoints_.size());
  hipDeviceSynchronize(); 
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "Time = " << elapsedTime << " ms\n";
}

void tps::CudaTPS::run() {
	allocResources();
	allocCudaResources();

  dim3 threadsPerBlock(32, 32);
  dim3 numBlocks(std::ceil(1.0*width/threadsPerBlock.x), std::ceil(1.0*height/threadsPerBlock.y));

  callKernel(cudaImageCoordCol, cudaSolutionCol, threadsPerBlock, numBlocks);
  callKernel(cudaImageCoordRow, cudaSolutionRow, threadsPerBlock, numBlocks);

  // std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  cudaRegistredImage<<<numBlocks, threadsPerBlock>>>(cudaImageCoordCol, cudaImageCoordRow, cudaImage, cudaRegImage, width, height);
  hipMemcpy(regImage, cudaRegImage, width*height*sizeof(uchar), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  std::cout << "Time = " << elapsedTime << " ms\n";

  registredImage.setPixelVector(regImage);
  registredImage.save(outputName_);

	freeResources();
	freeCudaResources();

	hipDeviceReset();
}

void tps::CudaTPS::allocResources() {
  regImage = (uchar*)malloc(width*height*sizeof(uchar));
  for (int col = 0; col < width; col++)
    for (int row = 0; row < height; row++)
      regImage[col*height+row] = 0;
  createCudaSolution();
  createCudaKeyPoint();
}

void tps::CudaTPS::createCudaSolution() {
  std::vector<float> solutionCol;
  std::vector<float> solutionRow;
  cudalienarSolver.solveLinearSystems();
  cudaSolutionCol = cudalienarSolver.getCudaSolCol();
  cudaSolutionRow = cudalienarSolver.getCudaSolRow();
}

void tps::CudaTPS::createCudaKeyPoint() {
  floatKeyCol = (float*)malloc(targetKeypoints_.size()*sizeof(float));
  floatKeyRow = (float*)malloc(targetKeypoints_.size()*sizeof(float));
  for (uint i = 0; i < referenceKeypoints_.size(); i++) {
    floatKeyCol[i] = referenceKeypoints_[i].x;
    floatKeyRow[i] = referenceKeypoints_[i].y;
  }
}

void tps::CudaTPS::allocCudaResources() {
  hipMalloc(&cudaImageCoordCol, width*height*sizeof(double));
  hipMalloc(&cudaImageCoordRow, width*height*sizeof(double));

  hipMalloc(&cudaKeyCol, targetKeypoints_.size()*sizeof(float));
  hipMalloc(&cudaKeyRow, targetKeypoints_.size()*sizeof(float));
  hipMemcpy(cudaKeyCol, floatKeyCol, targetKeypoints_.size()*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaKeyRow, floatKeyRow, targetKeypoints_.size()*sizeof(float), hipMemcpyHostToDevice);

  hipMalloc(&cudaRegImage, width*height*sizeof(uchar));
  hipMalloc(&cudaImage, width*height*sizeof(uchar));
  hipMemcpy(cudaImage, targetImage_.getPixelVector(), width*height*sizeof(uchar), hipMemcpyHostToDevice);
}

void tps::CudaTPS::freeResources() {
  free(floatKeyCol);
  free(floatKeyRow);
}

void tps::CudaTPS::freeCudaResources() {
  hipFree(cudaImageCoordCol);
  hipFree(cudaImageCoordRow);
  cudalienarSolver.freeCuda();
  hipFree(cudaKeyCol);
  hipFree(cudaKeyRow);
	hipDeviceSynchronize();
}
